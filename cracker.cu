#include "hip/hip_runtime.h"
//
// Created by grzegorz on 12.01.2020.
//

#include <iostream>
#include <cstring>
#include <chrono>
#include "cuda_clion_hack.hpp"
#include "hashing_algorithms/include/MD5_cuda_cracker.cuh"
//#include "hashing_algorithms/include/MD5_cpu_cracker.h"

unsigned int calculateWorkingBufferLength(unsigned int wordLength) {
    unsigned int toAdd = 64 - (wordLength + 8) % 64;
    if (toAdd == 0) toAdd = 64;
    return wordLength + toAdd + 8;
}

int crack(int min_length, int max_length, unsigned char *digest);

inline unsigned char hexToInt(unsigned char a, unsigned char b) {
    a = a - '0' < 10 ? a - '0' : a - 'a' + 10;
    b = b - '0' < 10 ? b - '0' : b - 'a' + 10;
    return (a * 16) + b;
}

int main(int argc, char **argv) {

    char digest_hex[DIGEST_LENGTH * 2 + 1];
    unsigned char digest[DIGEST_LENGTH];
    int min = 0;
    int max = 0;
    if (argc >= 4) {
        min = atoi(argv[1]);
        max = atoi(argv[2]);
        strcpy(reinterpret_cast<char *>(&digest_hex), argv[3]);
    }

    for (int i = 0; i < DIGEST_LENGTH; i++) {
        digest[i] = hexToInt(digest_hex[2 * i], digest_hex[2 * i + 1]);
    }

    crack(min, max, digest);

}

int crack(int min_length, int max_length, unsigned char *digest) {

    min_length = min_length >= 2 ? min_length : 2;
    hipError_t errorCode;
    const char NOT_FOUND[] = "-";

    char *word = new char[max_length + 1];
    char *word_gpu;

    unsigned char *digest_gpu;
    if ((errorCode = hipMalloc((void **) &digest_gpu, DIGEST_LENGTH * sizeof(unsigned char))) != hipSuccess) {
        std::cout << "error during alloc memory for digest on GPU error code: " << hipGetErrorName(errorCode)
                  << std::endl;
        return 1;
    };

    hipMemcpy(digest_gpu, digest, sizeof(unsigned char) * DIGEST_LENGTH, hipMemcpyHostToDevice);

    for (int length = min_length; length <= max_length; length++) {
        if ((errorCode = hipMalloc((void **) &word_gpu, length * sizeof(char))) != hipSuccess) {
            std::cout << "error during alloc memory for digest on GPU error code: " << hipGetErrorName(errorCode)
                      << std::endl;
            return 1;
        };
        hipMemcpy(word_gpu, NOT_FOUND, sizeof(char) * (strlen(NOT_FOUND) + 1), hipMemcpyHostToDevice);

        int workingBufferLength = calculateWorkingBufferLength(length);

        std::cout << "checking word with length: " << length << std::endl;

        auto startKernel = std::chrono::high_resolution_clock::now();

        calculateHashSum << < 256, 256 >> > (digest_gpu, word_gpu, workingBufferLength, length);

        auto stopKernel = std::chrono::high_resolution_clock::now();

        if ((errorCode = hipDeviceSynchronize()) != hipSuccess) {
            std::cout << "error during Device Synchronize: " << hipGetErrorName(errorCode)
                      << std::endl;
            return 1;
        }
        hipMemcpy(word, word_gpu, sizeof(char) * length, hipMemcpyDeviceToHost);
        word[length] = '\0';

        auto durationKernel = std::chrono::duration_cast<std::chrono::milliseconds>(stopKernel - startKernel);

        std::cout << word << "\tin: " << durationKernel.count() << std::endl;

        hipFree(word_gpu);
    }

    hipFree(digest_gpu);
    delete[]word;

    return 0;
}