#include "hip/hip_runtime.h"
//
// Created by grzegorz on 15.12.2019.
//


#include "include/MD5cudaDigestGenerator.cuh"
#include <iostream>
#include <chrono>
#include <cstring>
#include <cmath>

std::string MD5cudaDigestGenerator::getAlgorithmName() {
    return "md5_cuda";
}

unsigned int MD5cudaDigestGenerator::getDigestLength() {
    return 16;
}

void MD5cudaDigestGenerator::generate() {
    unsigned char *digestGPU;
    char *wordsGPU;

    auto startLoad = std::chrono::high_resolution_clock::now();

    unsigned long int workingBufferLength = calculateWorkingBufferLength(length_to_gen);
    if (workingBufferLength > 256) {
        std::cout << "error workingBufferLength > 2000 " << std::endl;
        return;
    }
    hipError_t errorCode;

    unsigned int wordBufferLength = length_to_gen+4-length_to_gen%4;

    if ((errorCode = hipMalloc((void **) &digestGPU, sizeof(unsigned char) * n_to_gen * getDigestLength())) !=
        hipSuccess) {
        std::cout << "error during alloc memory for digest on GPU error code: " << hipGetErrorName(errorCode)
                  << std::endl;
        return;
    };
    if ((errorCode = hipMalloc(&wordsGPU, sizeof(char) * n_to_gen * wordBufferLength)) != hipSuccess) {
        std::cout << "error during alloc memory for words on GPU error code: " << hipGetErrorName(errorCode)
                  << std::endl;
        return;
    };

    char *words_tmp = new char[wordBufferLength * n_to_gen];
    for (unsigned int i = 0; i < n_to_gen; i++) {
        memcpy(words_tmp + i * wordBufferLength, words[i], sizeof(unsigned char) * length_to_gen);
    }

    hipMemcpy(wordsGPU, words_tmp, sizeof(unsigned char) * wordBufferLength * n_to_gen, hipMemcpyHostToDevice);
    delete[] words_tmp;

    auto stopLoad = std::chrono::high_resolution_clock::now();
    auto durationLoad = std::chrono::duration_cast<std::chrono::milliseconds>(stopLoad - startLoad);
    std::cout << "gpu data load in: " << durationLoad.count() << " milliseconds" << std::endl;


    auto startKernel = std::chrono::high_resolution_clock::now();

    unsigned int blockSize = 64;
    unsigned int gridSize = (unsigned int) ceil((float) n_to_gen / blockSize);
//    std::cout << "number of blocks: " << gridSize << "\t number of threads per block: " << blockSize << std::endl;

    MD5_cuda::calculateHashSum <<< gridSize, blockSize >>> (digestGPU, wordsGPU, workingBufferLength, length_to_gen, n_to_gen);

    errorCode = hipDeviceSynchronize();

    auto stopKernel = std::chrono::high_resolution_clock::now();
    std::cout << "kernel quit code: " << hipGetErrorName(errorCode) << std::endl;

    auto durationKernel = std::chrono::duration_cast<std::chrono::milliseconds>(stopKernel - startKernel);
    std::cout << "kernel end work in in: " << durationKernel.count() << " milliseconds <-----------------" << std::endl;

    auto startUnload = std::chrono::high_resolution_clock::now();

    unsigned char *digest_tmp = new unsigned char[n_to_gen * getDigestLength()];
    hipMemcpy(digest_tmp, digestGPU, sizeof(unsigned char) * getDigestLength() * n_to_gen, hipMemcpyDeviceToHost);

    digest = new unsigned char *[n_to_gen];
    digest[0] = new unsigned char [n_to_gen*getDigestLength()];
    for (unsigned int i = 0; i < n_to_gen; i++) {
        digest[i] = digest_tmp + i * getDigestLength();
//        digest[i] = new unsigned char[getDigestLength()];
//        memcpy(digest[i], digest_tmp + i * getDigestLength(), getDigestLength());
    }

//    delete[] digest_tmp;
    hipFree(digestGPU);
    hipFree(wordsGPU);
    auto stopUnload = std::chrono::high_resolution_clock::now();
    auto durationUnload = std::chrono::duration_cast<std::chrono::milliseconds>(stopUnload - startUnload);
    std::cout << "gpu data unload in: " << durationUnload.count() << " milliseconds" << std::endl;

    n = n_to_gen;
    length = length_to_gen;

}

unsigned int MD5cudaDigestGenerator::calculateWorkingBufferLength(unsigned int defaultWordLength) {
    unsigned int toAdd = 64 - (defaultWordLength + 8) % 64;
    if (toAdd == 0) toAdd = 64;
    return defaultWordLength + toAdd + 8;
}

bool MD5cudaDigestGenerator::needOneDimArray() {
    return true;
}
