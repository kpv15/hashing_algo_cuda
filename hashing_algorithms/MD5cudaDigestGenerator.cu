#include "hip/hip_runtime.h"
//
// Created by grzegorz on 15.12.2019.
//


#include "include/MD5cudaDigestGenerator.cuh"
#include <iostream>
#include <chrono>
#include <cstring>
#include <cmath>

std::string MD5cudaDigestGenerator::getAlgorithmName() {
    return "md5_cuda";
}

unsigned int MD5cudaDigestGenerator::getDigestLength() {
    return 16;
}

void MD5cudaDigestGenerator::generate() {
    unsigned char *digestGPU;
    char *wordsGPU;

    auto startLoad = std::chrono::high_resolution_clock::now();

    unsigned long int workingBufferLength = calculateWorkingBufferLength(length_to_gen);
    if (workingBufferLength > 2000) {
        std::cout << "error workingBufferLength > 2000 " << std::endl;
        return;
    }
    hipError_t errorCode;

    if ((errorCode = hipMalloc((void **) &digestGPU, sizeof(unsigned char) * n_to_gen * getDigestLength())) !=
        hipSuccess) {
        std::cout << "error during alloc memory for digest on GPU error code: " << hipGetErrorName(errorCode)
                  << std::endl;
        return;
    };
    if ((errorCode = hipMalloc(&wordsGPU, sizeof(char) * n_to_gen * length_to_gen)) != hipSuccess) {
        std::cout << "error during alloc memory for words on GPU error code: " << hipGetErrorName(errorCode)
                  << std::endl;
        return;
    };

    char *words_tmp = new char[length_to_gen * n_to_gen];
    for (unsigned int i = 0; i < n_to_gen; i++) {
        memcpy(words_tmp + i * length_to_gen, words[i], sizeof(unsigned char) * length_to_gen);
    }

    hipMemcpy(wordsGPU, words_tmp, sizeof(unsigned char) * length_to_gen * n_to_gen, hipMemcpyHostToDevice);
    delete[] words_tmp;

    auto stopLoad = std::chrono::high_resolution_clock::now();
    auto durationLoad = std::chrono::duration_cast<std::chrono::milliseconds>(stopLoad - startLoad);
    std::cout << "gpu data load in: " << durationLoad.count() << " milliseconds" << std::endl;

    auto startKernel = std::chrono::high_resolution_clock::now();
    unsigned int blockSize = 64;
    unsigned int gridSize = (unsigned int) ceil((float) n_to_gen / blockSize);
    std::cout << "number of blocks: " << gridSize << "\t number of threads per block: " << blockSize << std::endl;
    calculateHashSum <<< gridSize, blockSize >>> (digestGPU, wordsGPU, workingBufferLength, length_to_gen, n_to_gen);

    hipDeviceSynchronize();

    auto stopKernel = std::chrono::high_resolution_clock::now();
    auto durationKernel = std::chrono::duration_cast<std::chrono::milliseconds>(stopKernel - startKernel);
    std::cout << "kernel end work in in: " << durationKernel.count() << " milliseconds" << std::endl;

    auto startUnload = std::chrono::high_resolution_clock::now();

    digest = new unsigned char *[n_to_gen];
    unsigned char *digest_tmp = new unsigned char[n_to_gen * getDigestLength()];
    hipMemcpy(digest_tmp, digestGPU, sizeof(unsigned char) * getDigestLength() * n_to_gen, hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < n_to_gen; i++) {
        digest[i] = new unsigned char[getDigestLength()];
        memcpy(digest[i], digest_tmp + i * getDigestLength(), getDigestLength());
    }

    delete[] digest_tmp;
    hipFree(digestGPU);
    hipFree(wordsGPU);
    auto stopUnload = std::chrono::high_resolution_clock::now();
    auto durationUnload = std::chrono::duration_cast<std::chrono::milliseconds>(stopUnload - startUnload);
    std::cout << "gpu data unload in: " << durationLoad.count() << " milliseconds" << std::endl;

    n = n_to_gen;
    length = length_to_gen;
}

unsigned int MD5cudaDigestGenerator::calculateWorkingBufferLength(unsigned int defaultWordLength) {
    unsigned int toAdd = 64 - (defaultWordLength + 8) % 64;
    if (toAdd == 0) toAdd = 64;
    return defaultWordLength + toAdd + 8;
}
