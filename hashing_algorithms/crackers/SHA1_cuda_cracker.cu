#include "hip/hip_runtime.h"
#include <cstring>
#include <cstdint>
#include "include/SHA1_cuda_cracker.cuh"
#include "../../cuda_clion_hack.hpp"

struct block {
    uint32_t a;
    uint32_t b;
    uint32_t c;
    uint32_t d;
    uint32_t e;
};

__constant__ block DEFAULT_DIGEST_BUFFER = {
        0x67452301,
        0xEFCDAB89,
        0x98BADCFE,
        0x10325476,
        0xC3D2E1F0
};

__device__ uint32_t leftRotate(uint32_t x, uint32_t n) {
    return (x << n) | (x >> (32 - n));
}

__device__ uint32_t funI(const uint32_t b, const uint32_t c, const uint32_t d) {
    return b ^ c ^ d;
}

__device__ uint32_t funH(const uint32_t b, const uint32_t c, const uint32_t d) {
    return (b & c) | (b & d) | (c & d);
}

__device__ uint32_t funG(const uint32_t b, const uint32_t c, const uint32_t d) {
    return b ^ c ^ d;
}

__device__ uint32_t funF(const uint32_t b, const uint32_t c, const uint32_t d) {
    return (b & c) | ((~b) & d);
}

__device__ uint32_t swap_bits(uint32_t x) {
    uint8_t *ptr = reinterpret_cast<uint8_t *>(&x);
    return (ptr[3] << 0) | (ptr[2] << 8) | (ptr[1] << 16) | (ptr[0] << 24);
}

#define MAX_WORD_SIZE 10
#define MAX_WORKING_BUFFER_SIZE MAX_WORD_SIZE + 128

__device__ void fillWorkingBuffer(const char *word, uint32_t *workingBuffer, unsigned int workingBufferLength,
                                  unsigned int wordLength) {

    unsigned int i = 0, j;
    uint32_t *word_ptr = (uint32_t *) word;
    for (i = 0; i < wordLength / 4; i++)
        workingBuffer[i] = swap_bits(word_ptr[i]);

    uint32_t split_word = 0;
    for (j = 0; j < wordLength % 4; j++)
        ((uint8_t *) &split_word)[3 - j] = word[wordLength / 4 * 4 + j];
    ((uint8_t *) &split_word)[3 - j] = 0b10000000;

    workingBuffer[i] = split_word;
    i++;

    while (i < workingBufferLength - 2) {
        workingBuffer[i++] = 0;
    }

    uint64_t tmp = wordLength * 8;
    std::memcpy(workingBuffer + i++, (uint32_t *) &tmp + 1, sizeof(uint32_t));
    std::memcpy(workingBuffer + i++, (uint32_t *) &tmp, sizeof(uint32_t));

}

__global__ void
calculateHashSum(unsigned char *digest_g, char *words, int workingBufferLength, int lenght) {

    __shared__ uint32_t digest[DIGEST_LENGTH / 4];
    for (int i = threadIdx.x; i < DIGEST_LENGTH / 4; i += blockDim.x)
        digest[i] = reinterpret_cast<uint32_t *>(digest_g)[i];
    __syncthreads();

    unsigned char workingBuffer[MAX_WORKING_BUFFER_SIZE];
    //init working buffer

    workingBuffer[0] = threadIdx.x;
    workingBuffer[1] = blockIdx.x;

    int combinations = 1;
    for (int i = 0; i < lenght - 2; i++)
        combinations *= 256;

    unsigned int numberOfChunks = workingBufferLength / 16;

    for (long j = 0; j < combinations; j++) {

        memcpy(workingBuffer + 2, &j, (lenght - 2) * sizeof(unsigned char));

        block mdBuffer = DEFAULT_DIGEST_BUFFER;
        block stepBuffer;
        uint32_t temp;
        uint32_t w[80];


        uint64_t tmp = lenght * 8;
        std::memcpy(workingBuffer + workingBufferLength - 2, (uint32_t *) &tmp + 1, sizeof(uint32_t));
        std::memcpy(workingBuffer + workingBufferLength - 1, (uint32_t *) &tmp, sizeof(uint32_t));


        for (unsigned int chunkNum = 0; chunkNum < numberOfChunks; chunkNum++) {
            memcpy(w, workingBuffer + chunkNum * 16, 16 * sizeof(uint32_t));

            for (int i = 16; i <= 79; i++)
                w[i] = leftRotate(w[i - 3] ^ w[i - 8] ^ w[i - 14] ^ w[i - 16], 1);

            stepBuffer = mdBuffer;

#pragma unroll
            for (int i = 0; i <= 79; i++) {
                if (i <= 19)
                    temp = leftRotate(stepBuffer.a, 5) + funF(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0x5A827999;
                else if (i <= 39)
                    temp = leftRotate(stepBuffer.a, 5) + funG(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0x6ED9EBA1;
                else if (i <= 59)
                    temp = leftRotate(stepBuffer.a, 5) + funH(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0x8F1BBCDC;
                else
                    temp = leftRotate(stepBuffer.a, 5) + funI(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0xCA62C1D6;
                stepBuffer.e = stepBuffer.d;
                stepBuffer.d = stepBuffer.c;
                stepBuffer.c = leftRotate(stepBuffer.b, 30);
                stepBuffer.b = stepBuffer.a;
                stepBuffer.a = temp;
            }
            mdBuffer.a += stepBuffer.a;
            mdBuffer.b += stepBuffer.b;
            mdBuffer.c += stepBuffer.c;
            mdBuffer.d += stepBuffer.d;
            mdBuffer.e += stepBuffer.e;
        }

        if (mdBuffer.a == reinterpret_cast<uint32_t *>(digest)[0] &&
            mdBuffer.b == reinterpret_cast<uint32_t *>(digest)[1] &&
            mdBuffer.c == reinterpret_cast<uint32_t *>(digest)[2] &&
            mdBuffer.d == reinterpret_cast<uint32_t *>(digest)[3] &&
            mdBuffer.e == reinterpret_cast<uint32_t *>(digest)[4]) {
            memcpy(words, &workingBuffer, lenght * sizeof(char));
//            words[0] = '1';
//            words[1] = '2';
//            words[2] = '3';
//            words[3] = '4';
//            words[4] = '5';
            __syncthreads();
        }
    }
}
