#include "hip/hip_runtime.h"
#include <cstring>
#include <cstdint>
#include "include/SHA1_cuda_cracker.cuh"

struct block {
    uint32_t a;
    uint32_t b;
    uint32_t c;
    uint32_t d;
    uint32_t e;
};

__constant__ block DEFAULT_DIGEST_BUFFER = {
        0x67452301,
        0xEFCDAB89,
        0x98BADCFE,
        0x10325476,
        0xC3D2E1F0
};

__device__ uint32_t leftRotate(uint32_t x, uint32_t n) {
    return (x << n) | (x >> (32 - n));
}

__device__ uint32_t funI(const uint32_t b, const uint32_t c, const uint32_t d) {
    return b ^ c ^ d;
}

__device__ uint32_t funH(const uint32_t b, const uint32_t c, const uint32_t d) {
    return (b & c) | (b & d) | (c & d);
}

__device__ uint32_t funG(const uint32_t b, const uint32_t c, const uint32_t d) {
    return b ^ c ^ d;
}

__device__ uint32_t funF(const uint32_t b, const uint32_t c, const uint32_t d) {
    return (b & c) | ((~b) & d);
}

__device__ uint32_t swap_bits(uint32_t x) {
    uint8_t *ptr = reinterpret_cast<uint8_t *>(&x);
    return (ptr[3] << 0) | (ptr[2] << 8) | (ptr[1] << 16) | (ptr[0] << 24);
}

#define MAX_WORD_SIZE 10
#define MAX_WORKING_BUFFER_SIZE MAX_WORD_SIZE + 128

__global__ void calculateHashSum(unsigned char *digest_g, char *message, int workingBufferLength, int lenght, volatile bool *kernel_end) {
    __shared__ uint32_t digest[DIGEST_LENGTH / 4];
    for (int i = threadIdx.x; i < DIGEST_LENGTH / 4; i += blockDim.x)
        digest[i] = reinterpret_cast<uint32_t *>(digest_g)[i];
    __syncthreads();
    __shared__ bool done;
    __shared__ unsigned char workingBuffer[MAX_WORKING_BUFFER_SIZE];
    if (threadIdx.x == 0) {
        memset(workingBuffer, 0, workingBufferLength * 4);
        //init working buffer
        workingBuffer[lenght] = 0b10000000;

        uint64_t tmp = lenght * 8;
        uint32_t l = swap_bits(((uint32_t *) &tmp)[0]);
        uint32_t h = swap_bits(((uint32_t *) &tmp)[1]);
        memcpy(workingBuffer + workingBufferLength * 4 - 8, &h, sizeof(uint32_t));
        memcpy(workingBuffer + workingBufferLength * 4 - 4, &l, sizeof(uint32_t));
        done = false;

    }
    __syncthreads();

    unsigned int numberOfChunks = workingBufferLength / 16;

    do {
        uint32_t w[80];
        block mdBuffer = DEFAULT_DIGEST_BUFFER;
        block stepBuffer;
        uint32_t temp;

        for (unsigned int chunkNum = 0; chunkNum < numberOfChunks; chunkNum++) {

            if (chunkNum == 0) {
                uint32_t X0 = threadIdx.x + (256 * blockIdx.x) + (uint32_t )(reinterpret_cast<uint32_t *>(&workingBuffer)[0]);
                w[0] = swap_bits(X0);
#pragma unroll
                for (int i = 1; i < 16; i++)
                    w[i] = swap_bits(reinterpret_cast<uint32_t *>(&workingBuffer + chunkNum * 16)[i]);
            } else {
#pragma unroll
                for (int i = 0; i < 16; i++)
                    w[i] = swap_bits(reinterpret_cast<uint32_t *>(&workingBuffer + chunkNum * 16)[i]);
            }

#pragma unroll
            for (int i = 16; i <= 79; i++)
                w[i] = leftRotate(w[i - 3] ^ w[i - 8] ^ w[i - 14] ^ w[i - 16], 1);

            stepBuffer = mdBuffer;

#pragma unroll
            for (int i = 0; i <= 79; i++) {
                if (i <= 19)
                    temp = leftRotate(stepBuffer.a, 5) + funF(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0x5A827999;
                else if (i <= 39)
                    temp = leftRotate(stepBuffer.a, 5) + funG(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0x6ED9EBA1;
                else if (i <= 59)
                    temp = leftRotate(stepBuffer.a, 5) + funH(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0x8F1BBCDC;
                else
                    temp = leftRotate(stepBuffer.a, 5) + funI(stepBuffer.b, stepBuffer.c, stepBuffer.d) +
                           stepBuffer.e + w[i] + 0xCA62C1D6;
                stepBuffer.e = stepBuffer.d;
                stepBuffer.d = stepBuffer.c;
                stepBuffer.c = leftRotate(stepBuffer.b, 30);
                stepBuffer.b = stepBuffer.a;
                stepBuffer.a = temp;
            }
            mdBuffer.a += stepBuffer.a;
            mdBuffer.b += stepBuffer.b;
            mdBuffer.c += stepBuffer.c;
            mdBuffer.d += stepBuffer.d;
            mdBuffer.e += stepBuffer.e;
        }

        if (mdBuffer.a == reinterpret_cast<uint32_t *>(digest)[0] &&
            mdBuffer.b == reinterpret_cast<uint32_t *>(digest)[1] &&
            mdBuffer.c == reinterpret_cast<uint32_t *>(digest)[2] &&
            mdBuffer.d == reinterpret_cast<uint32_t *>(digest)[3] &&
            mdBuffer.e == reinterpret_cast<uint32_t *>(digest)[4]) {
            memcpy(message, &workingBuffer, lenght * sizeof(char));
            reinterpret_cast<uint32_t *>(message)[0] += (blockIdx.x * 256) | threadIdx.x;
            *kernel_end = true;
        }
        __syncthreads();

        if (!done && threadIdx.x == 0) {

            int i = 2;
            while (i < lenght)
                workingBuffer[i++]++;
            done = true;
            for (int i = 2; i < lenght; i++) {
                if (workingBuffer[i] != 0) {
                    done = false;
                }
            }
        }
        __syncthreads();

    } while (!(done||*kernel_end));
}